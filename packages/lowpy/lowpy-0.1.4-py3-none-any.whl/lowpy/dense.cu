
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void propagate(
    const   int                     I,
    const   int                     iteration,
    const   double  * __restrict__  x,
    const   double  * __restrict__  w,
    const   double  * __restrict__  b,
            double  * __restrict__  y,
            double  * __restrict__  z
){
    int v = blockIdx.x;
    int j = blockIdx.y;
    int J = gridDim.y;
    double sum = 0;
    int inputIdx = 0;
    if (iteration > -1){
        inputIdx = iteration * I;
    }
    for (int i = 0; i < I; i++) sum += x[inputIdx+i]*w[v*J*I+j*I+i];
    y[v*J+j] = sum + b[v*J+j];
    z[v*J+j] = 1/(1 + exp(-y[v*J+j]));
}
__global__ void backpropagate(
    const   int                     iteration,
    const   int                     label,
            double * __restrict__   dedz,
    const   double * __restrict__   z,
    const   int                     J_n,
            double * __restrict__   w_n,
    const   double * __restrict__   dedz_n,
    const   double * __restrict__   dzdy_n,
            double * __restrict__   dzdy,
    const   double * __restrict__   alpha,
    const   int                     I,
            double * __restrict__   b,
            double * __restrict__   w,
    const   double * __restrict__   x,
    const   double * __restrict__   beta,
            double * __restrict__   mb,
            double * __restrict__   mw
){
    int v   = blockIdx.x;
    int j   = blockIdx.y;
    int J   = gridDim.y;
    int I_n = gridDim.y;
    int inputIdx = 0;
    if (iteration > -1){
        inputIdx = iteration * I;
    }
    if (label > -1){
        if (j == label){
            dedz[v*J+j] = z[v*J+j] - 1;
        }else{
            dedz[v*J+j] = z[v*J+j] - 0;
        }
    }else{
        double sum = 0;
        for (int j_n = 0; j_n < J_n; j_n++) sum += w_n[v*J_n*I_n+j+j_n*I_n] * dedz_n[v*J_n+j_n] * dzdy_n[v*J_n+j_n];
        dedz[v*J+j] = sum;
    }
    dzdy[v*J+j] = z[v*J+j] * (1 - z[v*J+j]);
    // b[l*J+j]   -= (beta[l] * mb[l*J+j] + alpha[l] * dedz[l*J+j] * dzdy[l*J+j]);
    // mb[l*J+j]  = (beta[l] * mb[l*J+j] + alpha[l] * dedz[l*J+j] * dzdy[l*J+j]);
    mb[v*J+j]  = beta[v] * mb[v*J+j] + alpha[v] * dedz[v*J+j] * dzdy[v*J+j];
    b[v*J+j]   -= mb[v*J+j];
    

    for (int i = 0; i < I; i++){
        //w[l*J*I+j*I+i]     -= (beta[l] * mw[l*J*I+j*I+i] + alpha[l] * dedz[l*J+j] * dzdy[l*J+j] * x[inputIdx+i]);
        //mw[l*J*I+j*I+i]     = (beta[l] * mw[l*J*I+j*I+i] + alpha[l] * dedz[l*J+j] * dzdy[l*J+j] * x[inputIdx+i]);
        mw[v*J*I+j*I+i]   = beta[v] * mw[v*J*I+j*I+i] + alpha[v] * dedz[v*J+j] * dzdy[v*J+j] * x[inputIdx+i];
        w[v*J*I+j*I+i]    -= mw[v*J*I+j*I+i];
        
    }
}
__global__ void argmax(
        const   int                     label,
        const   double  * __restrict__  z,
                int     * __restrict__  hits
){
    int v = blockIdx.x;
    int j = blockIdx.y;
    int J = gridDim.y;
    if (j == 0){
        double maxVal = 0;
        int maxIdx = 0;
        for (int i = 0; i < J; i++){
            if (z[v*J+i] > maxVal){
                maxIdx = i;
                maxVal = z[v*J+i];
            }
        }
        if (maxIdx == label){
            hits[v] += 1;
        }
    }
}